#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

void hello_cuda()
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    CHECK(hipDeviceReset());

#ifndef DLIB_USE_CUDA
#error why is this not defined?
#endif

    auto x = 4;
}

