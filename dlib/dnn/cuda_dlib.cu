#include "hip/hip_runtime.h"
// Copyright (C) 2015  Davis E. King (davis@dlib.net)
// License: Boost Software License   See LICENSE.txt for the full license.

#include "cuda_utils.h"
#include "cuda_dlib.h"


namespace dlib 
{ 
    namespace cuda 
    {

    // -----------------------------------------------------------------------------------

        void set_device (
            int dev
        )
        {
            CHECK_CUDA(hipSetDevice(dev));
        }

        int get_device (
        )
        {
            int dev = 0;
            CHECK_CUDA(hipGetDevice(&dev));
            return dev;
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_multiply(float* d, const float* s, size_t n)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] *= s[i];
            }
        }

        void multiply (
            tensor& dest,
            const tensor& src
        )
        {
            DLIB_CASSERT(dest.size()==src.size(),"");
            _cuda_multiply<<<512,512>>>(dest.device(), src.device(), src.size());
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform(float* d, const float* s, size_t n, float A, float B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s[i] + B;
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src,
            const float A,
            const float B
        )
        {
            DLIB_CASSERT(dest.size()==src.size(),"");
            _cuda_affine_transform<<<512,512>>>(dest.device(), src.device(), src.size(), A, B);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform(float* d, const float* s1, const float* s2, size_t n, float A, float B, float C)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i] + C;
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const float A,
            const float B,
            const float C
        )
        {
            DLIB_CASSERT(dest.size()==src1.size(),"");
            DLIB_CASSERT(dest.size()==src2.size(),"");
            _cuda_affine_transform<<<512,512>>>(dest.device(), src1.device(), src2.device(), dest.size(), A, B, C);
        }

    // ----------------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform(
            float* d, const float* s1, const float* s2, const float* s3, size_t n, float A, float B, float C, float D
        )
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A*s1[i] + B*s2[i] + C*s3[i] + D;
            }
        }

        void affine_transform(
            tensor& dest,
            const tensor& src1,
            const tensor& src2,
            const tensor& src3,
            const float A,
            const float B,
            const float C,
            const float D
        )
        {
            DLIB_CASSERT(dest.size()==src1.size(),"");
            DLIB_CASSERT(dest.size()==src2.size(),"");
            DLIB_CASSERT(dest.size()==src3.size(),"");
            _cuda_affine_transform<<<512,512>>>(dest.device(), src1.device(),
                src2.device(), src3.device(), dest.size(), A, B, C, D);
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_affine_transform2(float* d, const float* s, size_t n, const float* A, const float* B)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A[i]*s[i] + B[i];
            }
        }
        __global__ void _cuda_affine_transform3(float* d, const float* s, size_t n, const float* A, const float* B, size_t bs)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = A[i%bs]*s[i] + B[i%bs];
            }
        }

        void affine_transform(
            resizable_tensor& dest,
            const tensor& src,
            const tensor& A,
            const tensor& B
        )
        {
            DLIB_CASSERT(
                  ((A.num_samples()==1 && B.num_samples()==1) ||
                  (A.num_samples()==src.num_samples() && B.num_samples()==src.num_samples())) &&
                  A.nr()==B.nr() && B.nr()==src.nr() &&
                  A.nc()==B.nc() && B.nc()==src.nc() &&
                  A.k() ==B.k()  && B.k()==src.k(),"");

            dest.copy_size(src);
            if (A.num_samples() == 1)
            {
                _cuda_affine_transform3<<<512,512>>>(dest.device(), src.device(), src.size(), A.device(), B.device(), A.size());
            }
            else
            {
                _cuda_affine_transform2<<<512,512>>>(dest.device(), src.device(), src.size(), A.device(), B.device());
            }
        }

    // -----------------------------------------------------------------------------------

        void batch_normalize (
            resizable_tensor& dest,
            resizable_tensor& means,
            resizable_tensor& invstds,
            const tensor& src,
            const tensor& gamma, 
            const tensor& beta 
        )
        {
            // TODO
            DLIB_CASSERT(false,"");
        }

        void batch_normalize_gradient::operator() (
            const tensor& gradient_input,
            const tensor& means,
            const tensor& invstds,
            const tensor& src,
            const tensor& gamma,
            tensor& src_grad,
            tensor& gamma_grad, 
            tensor& beta_grad 
        )
        {
            // TODO
            DLIB_CASSERT(false,"");
        }

    // ----------------------------------------------------------------------------------------

        void batch_normalize_conv (
            resizable_tensor& dest,
            resizable_tensor& means,
            resizable_tensor& invstds,
            const tensor& src,
            const tensor& gamma, 
            const tensor& beta 
        )
        {
            // TODO
            DLIB_CASSERT(false,"");
        }

        void batch_normalize_conv_gradient::operator() (
            const tensor& gradient_input,
            const tensor& means,
            const tensor& invstds,
            const tensor& src,
            const tensor& gamma,
            tensor& src_grad,
            tensor& gamma_grad, 
            tensor& beta_grad 
        )
        {
            // TODO
            DLIB_CASSERT(false,"");
        }

    // -----------------------------------------------------------------------------------

        __global__ void _cuda_threshold(float* d, size_t n, float thresh)
        {
            for (auto i : grid_stride_range(0, n))
            {
                d[i] = d[i]>thresh ? 1:0;
            }
        }

        void threshold (
            tensor& data,
            float thresh
        )
        {
            _cuda_threshold<<<512,512>>>(data.device(), data.size(), thresh);
        }

    // ------------------------------------------------------------------------------------

    }
}

